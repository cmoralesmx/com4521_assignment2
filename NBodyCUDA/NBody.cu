#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <math.h>

#include "NBody.h"
#include "NBodyVisualiser.h"

#define USER_NAME "acp15cam"		//replace with your username
#define BUFFER_SIZE 512
#define SOFTENING_2 4.0f
#define DEBUG 0
#define ZERO 1e-6f
#define THREADS_PER_BLOCK 512
#define THREADS_PER_BLOCK2 64
#define THREADS_PER_BLOCK2_2 4096
#define WARP_SIZE 32

struct nbodies{
	float *x, *y, *vx, *vy, *m, *inv_m;
};
struct nbodies h_nbodies, d_nbodies;

void print_help();
void simulate(int iterations);
void step(void);
short operation_mode(const int argc, char *argv[]);
int fileReader(const char *filename);
int readLine(char buffer[], FILE *f);
char* copyString(const char * source);
short prepareData(const char * inputFilename);
char * getFilename(int argc, char *argv[], int secondValidCount, int secondPosition);
void assignDefaultValues(nbody *row);
void assignDefaultValuesSOA(int i);
void generateRandomData();
void displayData();
__global__ void parallelOverBodies(nbodies d_nbodies, float * activityMap, const int numberOfBodies, const float gridLimit, const unsigned short gridDimmension);
short allocateDeviceMemory();
__global__ void updateActivityMap(float * activityMap, const float inverse_numberOfBodies, const unsigned short gridDimmension, const unsigned short n2);
__global__ void body2body(float l_x, float l_y, float * s_x, float * s_y, float * s_vx, float * s_vy);
__global__ void sum_warp_kernel_shfl_down(float *a);
__global__ void parallelBody2Body(nbodies d_nbodies, float * d_activityMap, const int numberOfBodies, const float inv_gridLimit, const unsigned short gridDimm, const dim3 blocksPerGrid, const dim3 threadsPerBlock);
MODE mode;
int numberOfBodies;
float inverse_numberOfBodies;
short gridDimmension;
float * activityMap, gridLimit, *d_activityMap;
time_t t;


int main(int argc, char *argv[]) {
	int iterations;
	float seconds = -1.0f;
	
	srand((unsigned)time(NULL));
	clock_t begin = clock(), end = clock();

	// Check the received parameters follow the stablished format
	// and find if this is a simulation or visualisation
	short opMode = operation_mode(argc - 1, argv + 1);

	if ( opMode == -1 ){
		printf("Wrong parameters provided\n");
		print_help();
		return 0;
	}
	else {
		numberOfBodies = atoi(argv[1]); // N
		inverse_numberOfBodies = 1.0f / numberOfBodies;
		gridDimmension = atoi(argv[2]); // D
		// calculate the ranges for the "bins" of the grid
		gridLimit = 1.0f / (gridDimmension - 1);

		if (DEBUG){
			printf("Number of bodies: %d\n", numberOfBodies);
			printf("Activity grid dimmension: %d\n", gridDimmension);
		}

		// Allocate heap memory
		//data = (nbody*)malloc(sizeof(nbody) * numberOfBodies);
		size_t size = sizeof(float) * numberOfBodies;
		h_nbodies.x = (float*)malloc(size);
		h_nbodies.y = (float*)malloc(size);
		h_nbodies.vx = (float*)malloc(size);
		h_nbodies.vy = (float*)malloc(size);
		h_nbodies.m = (float*)malloc(size);
		h_nbodies.inv_m = (float*)malloc(size);
		activityMap = (float*)malloc(sizeof(float) * gridDimmension * gridDimmension);
		for (int v = 0; v < gridDimmension * gridDimmension; v++)
			activityMap[v] = 0;

		if (stricmp(argv[3], "CPU") == 0)
			mode = CPU;
		else if
			(stricmp(argv[3], "OPENMP") == 0)
				mode = OPENMP;
		else
		{
			mode = CUDA;
			allocateDeviceMemory();
		}

		if (DEBUG)
			printf("%s, %s mode\n", opMode == 0 ? "Simulation" : "Visualization", argv[3]);
		if (opMode == 0) {
			// simulation
			iterations = atoi(argv[5]);
			if (DEBUG)
				printf("Iterations: %d\n", iterations);
			// prepare simulation data
			// input parameters for simulation allow the filename to be specified in possition 7 for 8 params
			if (prepareData(getFilename(argc, argv, 8, 7)) == 0){
				if (DEBUG)
					printf("Simulating... ");
				
				// perform a fixed number of simulation steps (then output the timing results)
				begin = clock();
				simulate(iterations);
				end = clock();
				seconds = (end - begin) / (float)CLOCKS_PER_SEC;
				
				if (DEBUG)
					printf("Done!\n");
			}
		}
		else {
			// prepare visualisation data
			// input parameters for visualisation allow the filename to be specified in possition 5 for 6 params
			if (prepareData(getFilename(argc, argv, 6, 5)) == 0) {
				// configure and start the visualiser (then output the timing results).
				initViewer(numberOfBodies, gridDimmension, mode, &step);
				if (mode == CUDA){
					setNBodyPositions2f(d_nbodies.x, d_nbodies.y);
					setHistogramData(d_activityMap);
				}
				else{
					setNBodyPositions2f(h_nbodies.x, h_nbodies.y);
					setHistogramData(activityMap);
				}
				begin = clock();
				startVisualisationLoop();
				end = clock();
				seconds = (end - begin) / (float)CLOCKS_PER_SEC;
			}
		}
		if (seconds > -1.0f)
			printf("Execution time %.0f seconds %03.0f milliseconds\n", seconds, (seconds - (int)seconds) * 1000);
		else
			printf("No computation performed?");
		//getchar();*/
	}
	// release heap memory
	free(activityMap);
	free(h_nbodies.x);
	free(h_nbodies.y);
	free(h_nbodies.vx);
	free(h_nbodies.vy);
	free(h_nbodies.m);
	free(h_nbodies.inv_m);
	if (mode == CUDA){
		hipDeviceReset();
		hipFree(&d_nbodies.x);
		hipFree(&d_nbodies.y);
		hipFree(&d_nbodies.vx);
		hipFree(&d_nbodies.vy);
		hipFree(&d_nbodies.m);
		hipFree(&d_nbodies.inv_m);
		hipFree(&d_activityMap);
	}
	
	return 0;
}

short allocateDeviceMemory(){
	// allocate device dynamic global memory
	hipError_t cudaStatus1 = hipMalloc((void **)&d_nbodies.x, sizeof(d_nbodies.x) * numberOfBodies);
	hipError_t cudaStatus2 = hipMalloc((void **)&d_nbodies.y, sizeof(d_nbodies.y) * numberOfBodies);
	hipError_t cudaStatus3 = hipMalloc((void **)&d_nbodies.vx, sizeof(d_nbodies.vx) * numberOfBodies);
	hipError_t cudaStatus4 = hipMalloc((void **)&d_nbodies.vy, sizeof(d_nbodies.vy) * numberOfBodies);
	hipError_t cudaStatus5 = hipMalloc((void **)&d_nbodies.m, sizeof(d_nbodies.m) * numberOfBodies);
	hipError_t cudaStatus6 = hipMalloc((void **)&d_nbodies.inv_m, sizeof(d_nbodies.inv_m) * numberOfBodies);
	if (cudaStatus1 == hipSuccess && cudaStatus2 == hipSuccess && cudaStatus3 == hipSuccess
		&& cudaStatus4 == hipSuccess && cudaStatus5 == hipSuccess && cudaStatus6 == hipSuccess)
	{
		hipError_t cudaStatus = hipMalloc((void**)&d_activityMap, sizeof(float) * gridDimmension * gridDimmension);
		if (cudaStatus == hipSuccess)
		{
			cudaStatus = hipMemcpy(d_activityMap, activityMap, sizeof(float) * gridDimmension * gridDimmension, hipMemcpyHostToDevice);
			if (cudaStatus != hipSuccess){
				printf("Error copying the activityMap data to device memory");
				return 1;
			}
		}
		else{
			printf("Error allocating CUDA device memory for the activityMap");
			return 1;
		}
	}
	else{
		printf("Error allocating CUDA device memory for the data");
		return 1;
	}
	return 0;
}
char * getFilename(int argc, char *argv[], int secondValidCount, int secondPosition){
	if (argc == 7){
		return copyString(argv[6]);
	}
	else if (argc == secondValidCount){
		return copyString(argv[secondPosition]);
	}
	return NULL;
}

/*
input
const char * inputFilename - full path to the input file
output
-1	a line read does not match the required format
1	the number of records and number of bodies does not match
0	process completed successfully
*/
short loadOrGenerateData(const char * inputFilename){
	if (inputFilename != NULL){
		// read data from file
		return fileReader(inputFilename);
	}
	else {
		// generate random data.
		generateRandomData();
		return 0;
	}
}

/*
input 
const char * inputFilename - full path to the input file
output
-1	a line read does not match the required format
1	the number of records and number of bodies does not match
0	process completed successfully
-2	CUDA error copying data to device
*/
short prepareData(const char * inputFilename){
	short loadDataErrors = loadOrGenerateData(inputFilename);
	if (loadDataErrors != 0){
		if (DEBUG)
			printf("ERROR loading or generating data!\n");
	}
	if (mode == CUDA){
		// copy host data to device
		hipError_t cudaStatus1, cudaStatus2, cudaStatus3, cudaStatus4, cudaStatus5, cudaStatus6;
		size_t size = sizeof(float) * numberOfBodies;
		cudaStatus1 = hipMemcpy(d_nbodies.x, h_nbodies.x, size, hipMemcpyHostToDevice);
		cudaStatus2 = hipMemcpy(d_nbodies.y, h_nbodies.y, size, hipMemcpyHostToDevice);
		cudaStatus3 = hipMemcpy(d_nbodies.vx, h_nbodies.vx, size, hipMemcpyHostToDevice);
		cudaStatus4 = hipMemcpy(d_nbodies.vy, h_nbodies.vy, size, hipMemcpyHostToDevice);
		cudaStatus5 = hipMemcpy(d_nbodies.m, h_nbodies.m, size, hipMemcpyHostToDevice);
		cudaStatus6 = hipMemcpy(d_nbodies.inv_m, h_nbodies.inv_m, size, hipMemcpyHostToDevice);
		if (cudaStatus1 != hipSuccess || cudaStatus2 != hipSuccess || cudaStatus3 != hipSuccess
			|| cudaStatus4 != hipSuccess || cudaStatus5 != hipSuccess || cudaStatus6 != hipSuccess){
			if (DEBUG)
				printf("ERROR copying host data to device\n");
			return -2;
		}
	}
	return loadDataErrors;
}

void displayData(){
	for (int i = 0; i < numberOfBodies; i++){
		if (numberOfBodies < 10 || i < 5 || i > numberOfBodies - 5)
			printf("[%d] x=%f, y=%f, vx=%f, vy=%f, mass=%f\n", i, h_nbodies.x[i],
				h_nbodies.y[i], h_nbodies.vx[i], h_nbodies.vy[i], h_nbodies.m[i]);
	}
}
/* function to simplify memory allocation and content copy
	taken from C p.88
	Tried to implement the pointer version of strcopy from the book but does
	not work so I had to resort to the subscript version
*/
char* copyString(const char * source){
	int i = 0;
	char * dest = (char *)malloc(strlen(source) * sizeof(char));
	
	//while (*dest++ = *source++); // this method does not work, why?
	while (dest[i] = source[i])
		i++;
	return dest;
}

void simulate(int iterations)
{
	for (int iteration = 0; iteration < iterations; iteration++){
		step();
	}
}

/*
Perform a simulation step of the system
*/
void step(void)
{
	int i, j;

	switch (mode)
	{
	case CPU:
		for (i = 0; i < numberOfBodies; i++)
		{
			float euclidean_x, euclidean_y, soft_norm, force_x, force_y;
			float sum_x = 0, sum_y = 0;

			for (j = 0; j < numberOfBodies; j++){
				// m_j (x_j - x_i) / (|| x_j - x_i ||^2 + softening^2 )^(3/2)
				euclidean_x = h_nbodies.x[j] - h_nbodies.x[i];
				euclidean_y = h_nbodies.y[j] - h_nbodies.y[i];
				soft_norm = (float)pow(euclidean_x * euclidean_x + euclidean_y * euclidean_y + SOFTENING_2, 1.5f) + ZERO;
				// this simation is independent for x or y
				sum_x += (h_nbodies.m[j] * euclidean_x) / soft_norm;
				sum_y += (h_nbodies.m[j] * euclidean_y) / soft_norm;
			}
			// Calculate the force
			// F_i = G * m_i * sum
			force_x = G * h_nbodies.m[i] * sum_x;
			force_y = G * h_nbodies.m[i] * sum_y;

			// simulate the movement

			// calculate the position
			// WE DO THIS FIRST due to its dependance on current velocity
			// x_t+1 = x_t + dt * v_t
			h_nbodies.x[i] += dt * h_nbodies.vx[i];
			h_nbodies.y[i] += dt * h_nbodies.vy[i];

			// update the velocity value 
			// acceleration is also computed here, no need for independent computation
			// v_t+1 = v_t + dt * a  // acceleration a_i = F_i / m_i
			h_nbodies.vx[i] += dt * (force_x / (h_nbodies.m[i] + ZERO));
			h_nbodies.vy[i] += dt * (force_y / (h_nbodies.m[i] + ZERO));

			/*
			compute the position for a body in the activityMap and increase the
			corresponding body count
			index computed according to "The C programming guide" 2nd ed pp.113
			*/
			int col = (int)(h_nbodies.x[i] / (gridLimit + ZERO));
			int row = (int)(h_nbodies.y[i] / (gridLimit + ZERO));
			int cell = (int)(gridDimmension * row + col);
			activityMap[cell] += 1.0f;
		}
		// Now traverse the activityMap to normalise the counts
		// to achieve the intended visualization
		for (i = 0; i < gridDimmension * gridDimmension; i++)
		{
			activityMap[i] /= (float)numberOfBodies;
			activityMap[i] *= gridDimmension;
		}
		break;
	case OPENMP:
//#pragma omp parallel num_threads(4)
	{
		//#pragma omp parallel for default(none) shared(data, activityMap, numberOfBodies, gridLimit, gridDimmension)
		for (i = 0; i < numberOfBodies; i++)
		{
			float euclidean_x, euclidean_y, soft_norm, force_x, force_y;
			float sum_x = 0, sum_y = 0;

#pragma omp parallel for reduction(+: sum_x, sum_y) default(none) shared(h_nbodies, activityMap, numberOfBodies, i, gridLimit, gridDimmension) private (euclidean_x, euclidean_y, soft_norm)
			for (j = 0; j < numberOfBodies; j++)
			{
				// m_j (x_j - x_i) / (|| x_j - x_i ||^2 + softening^2 )^(3/2)
				euclidean_x = h_nbodies.x[j] - h_nbodies.x[i];
				euclidean_y = h_nbodies.y[j] - h_nbodies.y[i];
				soft_norm = (float)pow(euclidean_x * euclidean_x + euclidean_y * euclidean_y + SOFTENING_2, 1.5f) + ZERO;
				// this simation is independent for x or y
				sum_x += (h_nbodies.m[j] * euclidean_x) / soft_norm;
				sum_y += (h_nbodies.m[j] * euclidean_y) / soft_norm;
			}
			// Calculate the force
			// F_i = G * m_i * sum
			force_x = G * h_nbodies.m[i] * sum_x;
			force_y = G * h_nbodies.m[i] * sum_y;

			// simulate the movement

			// calculate the position
			// WE DO THIS FIRST due to its dependance on current velocity
			// x_t+1 = x_t + dt * v_t
			h_nbodies.x[i] += dt * h_nbodies.vx[i];
			h_nbodies.y[i] += dt * h_nbodies.vy[i];

			// update the velocity value 
			// acceleration is also computed here, no need for independent computation
			// v_t+1 = v_t + dt * a  // acceleration a_i = F_i / m_i
			h_nbodies.vx[i] += dt * (force_x / (h_nbodies.m[i] + ZERO));
			h_nbodies.vy[i] += dt * (force_y / (h_nbodies.m[i] + ZERO));

			/*
			compute the position for a body in the activityMap and increase the
			corresponding body count
			index computed according to "The C programming guide" 2nd ed pp.113
			*/
			int col = (int)(h_nbodies.x[i] / (gridLimit + ZERO));
			int row = (int)(h_nbodies.y[i] / (gridLimit + ZERO));
			int cell = (int)(gridDimmension * row + col);
#pragma omp atomic
			activityMap[cell] += 1.0f;
		}
#pragma omp parallel for schedule(dynamic) default(none) shared(gridDimmension, activityMap, numberOfBodies)
		for (i = 0; i < gridDimmension * gridDimmension; i++){
			activityMap[i] /= (float)numberOfBodies;
			activityMap[i] *= gridDimmension;
		}
	}
		break;
	case CUDA:
		// launch the bodies kernel
		dim3 blocksPerGrid((numberOfBodies + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1);
		dim3 threadsPerBlock(THREADS_PER_BLOCK, 1);
		
		// First CUDA option
		//parallelOverBodies << < blocksPerGrid, threadsPerBlock >> >(d_nbodies, d_activityMap, numberOfBodies, 1.0f/gridLimit, gridDimmension);
		// second CUDA option
		parallelBody2Body << <blocksPerGrid, threadsPerBlock >> >(d_nbodies, activityMap, numberOfBodies, 1.0f / gridLimit, gridDimmension, blocksPerGrid, threadsPerBlock);
		hipError_t cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess)
			printf("CUDA error in bodies kernel\n");
		
		// sumation over shared
		// pre-update activity matrix???

		// launch the activity map updater kernel
		updateActivityMap << < blocksPerGrid, threadsPerBlock >> >(d_activityMap, inverse_numberOfBodies, gridDimmension, gridDimmension * gridDimmension);
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess)
			printf("CUDA error in activity map kernel\n");

		break;
	}
}
__global__ void updateActivityMap(float * d_activityMap, const float inverse_numberOfBodies, const unsigned short gridDimmension, const unsigned short n2){
	unsigned short idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < n2){
		d_activityMap[idx] *= inverse_numberOfBodies;
		d_activityMap[idx] *= gridDimmension;
	}
}
/*
parallelOverBodies - The kernel computes the affecting forces per body.
*/
__global__ void parallelOverBodies(nbodies d_nbodies, float * d_activityMap, const int numberOfBodies, const float inv_gridLimit, const unsigned short gridDimm){
	unsigned short idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < numberOfBodies){
		float4 force = { 0.0f, 0.0f, 0.0f, 0.0f };
		nbody body = { d_nbodies.x[idx], d_nbodies.y[idx], d_nbodies.vx[idx], d_nbodies.vy[idx], d_nbodies.m[idx] };
		for (short block = 0; block < gridDim.x; block++){
			__shared__ float s_x[THREADS_PER_BLOCK];
			__shared__ float s_y[THREADS_PER_BLOCK];
			__shared__ float s_m[THREADS_PER_BLOCK];
			unsigned short tid = block * blockDim.x + threadIdx.x;
			s_x[threadIdx.x] = d_nbodies.x[tid];
			s_y[threadIdx.x] = d_nbodies.y[tid];
			s_m[threadIdx.x] = d_nbodies.m[tid];
			__syncthreads();

			for (int j = 0; j < THREADS_PER_BLOCK; j++){
				// m_j (x_j - x_i) / (|| x_j - x_i ||^2 + softening^2 )^(3/2)
				float distance_x = s_x[j] - body.x;
				float distance_y = s_y[j] - body.y;
				
				float dx_2 = distance_x * distance_x;
				float dy_2 = distance_y * distance_y;
				float subt = dx_2 + dy_2;
				// CUDA reciprocal squared root. faster than 1/sqrt(x)
				float inv_sqrt = rsqrtf(subt + SOFTENING_2);
				float inv_sqrt_2 = inv_sqrt * inv_sqrt;
				float inv_sqrt_3 =  inv_sqrt_2 * inv_sqrt;
				// this sumation is independent for x or y
				float mass_inv_sqrt_3 = s_m[j] * inv_sqrt_3;
				force.z += mass_inv_sqrt_3 * distance_x;
				force.w += mass_inv_sqrt_3 * distance_y;
			}
			__syncthreads();
		}
		// Calculate the force
		// F_i = G * m_i * sum
		force.x = G * body.m * force.z;
		force.y = G * body.m * force.w;

		// simulate the movement

		// calculate the position
		// WE DO THIS FIRST due to its dependance on current velocity
		// x_t+1 = x_t + dt * v_t
		d_nbodies.x[idx] += dt * body.vx;
		d_nbodies.y[idx] += dt * body.vy;

		// update the velocity value 
		// acceleration is also computed here, no need for independent computation
		// v_t+1 = v_t + dt * a  // acceleration a_i = F_i / m_i
		d_nbodies.vx[idx] += dt * force.x * d_nbodies.inv_m[idx];
		d_nbodies.vy[idx] += dt * force.y * d_nbodies.inv_m[idx];

		/*
		compute the position for a body in the activityMap and increase the
		corresponding body count
		index computed according to "The C programming guide" 2nd ed pp.113
		*/
		unsigned short col = d_nbodies.x[idx] * inv_gridLimit;
		unsigned short row = d_nbodies.y[idx] * inv_gridLimit;
		unsigned short cell = gridDimm * row + col;
		
		atomicAdd(&d_activityMap[cell], 1.0f);
	}
}

/*
this kernel should compute, the amount of effect each body in matrix B has over the bodies in matrix A
s_accum_vx[padding + threadIdx.x]
s_accum_vy[padding + threadIdx.x]
*/
__global__ void body2body(float l_x, float l_y,
	float * s_x, float * s_y, 
	float * s_vx, float * s_vy){

//	unsigned short itid = blockIdx.x * blockDim.x + threadIdx.x;

	float distance_x = s_x[threadIdx.x] - l_x;
	float distance_y = s_y[threadIdx.x] - l_y;

	float dx_2 = distance_x * distance_x;
	float dy_2 = distance_y * distance_y;
	float subt = dx_2 + dy_2;
	// CUDA reciprocal squared root. faster than 1/sqrt(x)
	float inv_sqrt = rsqrtf(subt + SOFTENING_2);
	float inv_sqrt_3 = inv_sqrt * inv_sqrt * inv_sqrt;
	// this sumation is independent for x or y
	//float mass_inv_sqrt_3 = l_m * inv_sqrt_3;
	s_vx[threadIdx.x] = inv_sqrt_3 * distance_x;
	s_vy[threadIdx.x] = inv_sqrt_3 * distance_y;
}

__global__ void sum_warp_kernel_shfl_down(float *a)
{
	float local_sum = a[threadIdx.x + blockIdx.x * blockDim.x];
	for (int offset = WARP_SIZE / 2; offset>0; offset /= 2)
		local_sum += __shfl_down(local_sum, offset);
	if (threadIdx.x % WARP_SIZE == 0){
		//printf("Warp max is %d", local_sum);
		a[0] = local_sum;
	}
}

/*
parallelBody2Body - This kernel computes the body-to-body interactions.

*/
__global__ void parallelBody2Body(nbodies d_nbodies, float * d_activityMap, const int numberOfBodies, 
	const float inv_gridLimit, const unsigned short gridDimm, const dim3 blocksPerGrid, const dim3 threadsPerBlock){
	unsigned short tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid < numberOfBodies){
		// shared memory containers to use as the base for computation
		__shared__ float s_x[THREADS_PER_BLOCK];
		__shared__ float s_y[THREADS_PER_BLOCK];
		__shared__ float s_vx[THREADS_PER_BLOCK];
		__shared__ float s_vy[THREADS_PER_BLOCK];
		__shared__ float s_m[THREADS_PER_BLOCK];
		__shared__ float s_inv_m[THREADS_PER_BLOCK];
		// l_* myPosition
		//float l_x = d_nbodies.x[tid], l_y = d_nbodies.y[tid], l_m = d_nbodies.m[tid];
		// temp accumulator for the velocity
		float accum_vx = 0.0f;
		float accum_vy = 0.0f;

		// initiate te values
		s_x[threadIdx.x] = d_nbodies.x[tid];
		s_y[threadIdx.x] = d_nbodies.y[tid];
		s_m[threadIdx.x] = d_nbodies.m[tid];
		s_inv_m[threadIdx.x] = d_nbodies.inv_m[tid];
		for (int i = 0, tile = 0; i < numberOfBodies; i += THREADS_PER_BLOCK, tile++){
			//unsigned short idx = tile * blockDim.x + threadIdx.x;
			for (int z = 0; z < THREADS_PER_BLOCK; z++){
				s_vx[z] = 0.0f;
				s_vy[z] = 0.0f;
			}
			__syncthreads();
			// launch 1 block iteratively to compute the body 2 body interactions
			// should pass single values: s_x[tile], s_y[tile]
			// pointers to address to begin reading the array at: d_nbodies.x[tile * blockDim.x], d_nbodies.y[tile * blockDim.x]
			// pointers to array: s_vx, s_vy
			body2body << <1, threadsPerBlock >> >(s_x[threadIdx.x], s_y[threadIdx.x],
				&d_nbodies.x[tile * blockDim.x], &d_nbodies.x[tile * blockDim.x], s_vx, s_vy);
			__syncthreads();
			// shuffle warp sum the computed values for this block
			sum_warp_kernel_shfl_down << <blocksPerGrid, threadsPerBlock, 1 >> >(s_vx);
			sum_warp_kernel_shfl_down << <blocksPerGrid, threadsPerBlock, 2 >> >(s_vy);
			// accumulate the velocity values
			accum_vx += s_vx[0];
			accum_vy += s_vy[0];
		}
		// update the rest of the values
		// Calculate the force
		// F_i = G * m_i * sum
		float force_x = G * s_m[threadIdx.x] * accum_vx;
		float force_y = G * s_m[threadIdx.x] * accum_vy;

		// simulate the movement
		// these values should be output to the accumulator array for further reduction

		// calculate the new position for this body
		// x_t+1 = x_t + dt * v_t
		s_x[threadIdx.x] += dt * accum_vx;
		s_y[threadIdx.x] += dt * accum_vy;

		// update the velocity value 
		// acceleration is also computed here, no need for independent computation
		// v_t+1 = v_t + dt * a  // acceleration a_i = F_i / m_i
		s_vx[threadIdx.x] += dt * force_x * s_inv_m[tid]; // these seem to need sumation at the end
		s_vy[threadIdx.x] += dt * force_y * s_inv_m[tid]; // needs to be fixed?

		// save the values to global memory
		d_nbodies.x[tid] = s_x[threadIdx.x];
		d_nbodies.y[tid] = s_y[threadIdx.x];
		d_nbodies.vx[tid] = s_vx[threadIdx.x];
		d_nbodies.vy[tid] = s_vy[threadIdx.x];
		///*
		//compute the position for a body in the activityMap and increase the
		//corresponding body count
		//index computed according to "The C programming guide" 2nd ed pp.113
		//*/
		unsigned short col = s_x[threadIdx.x] * inv_gridLimit;
		unsigned short row = s_y[threadIdx.x] * inv_gridLimit;
		unsigned short cell = gridDimm * row + col;

		atomicAdd(&d_activityMap[cell], 1.0f);

	}
}
void print_help(){
	printf("nbody_%s N D M [-i I] [-i input_file]\n", USER_NAME);

	printf("where:\n");
	printf("\tN                Is the number of bodies to simulate.\n");
	printf("\tD                Is the integer dimension of the activity grid. The Grid has D*D locations.\n");
	printf("\tM                Is the operation mode, either  'CPU', 'OPENMP' or 'CUDA'\n");
	printf("\t[-i I]           Optionally specifies the number of simulation iterations 'I' to perform. \n\t\t\t\tSpecifying no value will use visualisation mode. \n");
	printf("\t[-f input_file]  Optionally specifies an input file with an initial N bodies of data. \n\t\t\t\tIf not specified random data will be created.\n");
}

/**
Validate the inputs provided in comliance with print_help function
input:
int argc
char *argv[]
returns:
-1 wrong parameters provided
1 parameters specify visualisation run
0 parameters specify simulation run
*/
short operation_mode(const int argc, char **argv){
	if (argc < 3 || argc > 7){
		return -1;
	}
	if (argc < 5){
		// This number of parameters can only match a visualisation run

		// check for integer values in first parameters
		if (atoi(argv[0]) <= 0 || atoi(argv[1]) <= 0)
			return -1;
		// parameter 3 only CPU or OPENMP valid
		if (stricmp(argv[2], "CPU") != 0 && stricmp(argv[2], "OPENMP") != 0 && stricmp(argv[2], "CUDA") != 0)
			return -1;

		// 5th parameter must be '-i', if present
		if (argc == 5 && stricmp(argv[4], "-i") != 0)
			return -1;

		// parameters seem to comply
		return 1;
	}
	else {
		// check for integer values in first parameters
		if (atoi(argv[0]) == 0 || atoi(argv[1]) == 0)
			return -1;
		// parameter 3 only CPU or OPENMP valid
		if (stricmp(argv[2], "CPU") != 0 && stricmp(argv[2], "OPENMP") != 0 && stricmp(argv[2], "CUDA") != 0)
			return -1;

		// for 6 params, there must be a numeric value in param 5
		// params -i -f can not be specified both w.o. parameters at the same time
		if (argc == 5)
			if (stricmp(argv[3], "-f") == 0)
				// iterations were skipped but the file was provided
				// this is an unlimited visualisation
				return 1;
			else
				if (atoi(argv[4]) == 0)
					return -1;
				else
					// this is a simulation with N iteration specified
					return 0;

		// for 7 params, the last one can only be the path to the input file
		if (argc == 6)
			if (stricmp(argv[5], "-f") == 0)
				return -1;
			else
				// this is a visualistation with an input file specified
				return 1;

		// Number of simulation cannot be 0
		if (argc == 7 && atoi(argv[4]) == 0)
			return -1;

		// this is a simulation with full parameters specified. correctly?
		return 0;
	}
}
/*
Counts the number of commas in the buffer
input:
const char * buffer - the buffer to use as source
output:
int the number of commas
*/
short countCommas(const char * buffer){
	unsigned int i;
	unsigned short commas = 0;
	// Check that the line contains 4 commas
	for (i = 0; i < strlen(buffer); i++){
		if (buffer[i] == ',')
			++commas;
	}
	return commas;
}
/*
Executes a loop to fill the data structure with default parameters
*/
void generateRandomData(){
	if (DEBUG)
		printf("Generating random data for %d bodies. ", numberOfBodies);
	for (int i = 0; i < numberOfBodies; i++)
		assignDefaultValuesSOA(i);
	if (DEBUG)
		printf("Done.\n");
}
/*
Assigns default values in accordance to specifications
x,y = random [0,1]
vx,vy = 0
mass = 1/N
input:
float* row - pointer to the array to fill
const int N - number of expected bodies
return:
void
*/
void assignDefaultValues(nbody *row){
	//(double)rand() / (double)((unsigned)RAND_MAX + 1)
	row->x = (float)((double)rand() / (double)((unsigned)RAND_MAX + 1));
	row->y = (float)((double)rand() / (double)((unsigned)RAND_MAX + 1));
	if (row->x < 0.000001)
		row->x = 0;
	if (row->y < 0.000001) 
		row->y = 0;
	row->vx = 0;
	row->vy = 0;
	row->m = 1.0f / (float)numberOfBodies;
}
void assignDefaultValuesSOA(int i){
	//(double)rand() / (double)((unsigned)RAND_MAX + 1)
	h_nbodies.x[i] = (float)((double)rand() / (double)((unsigned)RAND_MAX + 1));
	h_nbodies.y[i] = (float)((double)rand() / (double)((unsigned)RAND_MAX + 1));
	if (h_nbodies.x[i] < 0.000001)
		h_nbodies.x[i] = 0;
	if (h_nbodies.y[i] < 0.000001)
		h_nbodies.y[i] = 0;
	h_nbodies.vx[i] = 0;
	h_nbodies.vy[i] = 0;
	h_nbodies.m[i] = 1.0f / (float)numberOfBodies;
	h_nbodies.inv_m[i] = 1.0f / h_nbodies.m[i];
}
/*
Loads the data from specified input file.
Fills any value not provided with default values in accordance to specifications
input:
const char * filename - pointer to the file to read
const int N - number of bodies expected in the file
float ** data - pre allocated 2d container array for the data to loadS
output:
-1	a line read does not match the required format (4 commas)
1	the number of records in and number of bodies does not match
0	process completed successfully
*/
int fileReader(const char *filename){
	char buffer[BUFFER_SIZE];
	int body_count = 0;

	if (DEBUG)
		printf("Input file: %s\n", filename);

	FILE *f = fopen(filename, "r");

	while (readLine(buffer, f)){
		
		if (buffer[0] == '#') // comment lines are ignored
			continue;

		if (countCommas(buffer) != 4){
			// the line read does not follow the required format
			fclose(f);
			return -1;
		}

		assignDefaultValuesSOA(body_count);

		if (body_count < numberOfBodies){
			// valid format: 0.5f, 0.5f, 0.0f, 0.0f, 0.1f
			sscanf(buffer, "%ff, %ff, %ff, %ff, %ff", &h_nbodies.x[body_count],
				&h_nbodies.y[body_count], &h_nbodies.vx[body_count],
				&h_nbodies.vy[body_count], &h_nbodies.m[body_count]);
			h_nbodies.inv_m[body_count] = 1.0f / h_nbodies.m[body_count];
			++body_count;
		}
	}
	fclose(f);
	if (body_count != numberOfBodies){
		printf("Number of bodies in input file does not match the parameter specified\n");
		return 1;
	}
	return 0;
}
/* Reads a single line from the specified file
input:
char buffer[] - the buffer to store the read characters
const FILE *f - a pointer to the file to read
output:
0 upon reaching the EOF indicator
1 for a line successfully read
*/
int readLine(char buffer[], FILE *f){
	unsigned short i = 0;
	char c = 0;

	while ((c = getc(f)) != '\n'){
		if (c == EOF)
			return 0;
		// Add character to buffer
		buffer[i++] = c;
		// Check index for overflow
		if (i == BUFFER_SIZE){
			fprintf(stderr, "buffer overflow");
			exit(0);
		}
	}
	// Ensure the buffer is correctly terminated
	buffer[i] = '\0';
	
	return 1;
}